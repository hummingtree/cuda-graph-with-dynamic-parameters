#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2022, NVIDIA Corporation
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy of
 * this software and associated documentation files (the "Software"), to deal in
 * the Software without restriction, including without limitation the rights to
 * use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
 * the Software, and to permit persons to whom the Software is furnished to do so,
 * subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
 * FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
 * COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
 * IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 * CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

#include "gpu_graph.hpp"
#include "cuda_helper.hpp"

constexpr int n_kernel = 2;
constexpr int n_iteration = 40000;

__global__ void shortKernel(float *out_d, const float *in_d, int N, float f){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < N) { 
      out_d[idx] = f * in_d[idx];
  }
}

__global__ void initKernel(float *ptr, int N, float f){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < N) { 
      ptr[idx] = f;
  }
}

void run_kernels_graph(float *out_d, float *in_d, int size, float f, gpu_graph_t &g, hipStream_t s)
{
  constexpr int threads = 256;
  int blocks = (size + threads - 1) / threads;

  for(int i = 0; i < n_kernel; i++){
    hipKernelNodeParams params;
    params.blockDim = {static_cast<unsigned int>(threads), 1, 1};
    params.gridDim = {static_cast<unsigned int>(blocks), 1, 1};
    params.sharedMemBytes = 0;
    params.func = reinterpret_cast<void *>(shortKernel);
    void *args[] = {&out_d, &in_d, &size, &f};
    params.kernelParams = args;
    params.extra = nullptr;

    if (g.state() == gpu_graph_t::state_t::capture) {
      // Static kernels
      shortKernel<<<blocks, threads, 0, s>>>(out_d, in_d, size, 1.004f);
      shortKernel<<<blocks, threads, 0, s>>>(in_d, out_d, size, 1.004f);

      // kernels with dynamic parameter `f`
      // Add the kernel nodes
      g.add_kernel_node(i * 2 + 0, params, s);
      params.kernelParams[0] = &in_d;
      params.kernelParams[1] = &out_d;
      g.add_kernel_node(i * 2 + 1, params, s);
    } else if (g.state() == gpu_graph_t::state_t::update) {
      // Update the kernel nodes
      g.update_kernel_node(i * 2 + 0, params);
      params.kernelParams[0] = &in_d;
      params.kernelParams[1] = &out_d;
      g.update_kernel_node(i * 2 + 1, params);
    }
  } 
}

void run_kernels_no_graph(float *out_d, float *in_d, int size, float f, hipStream_t s)
{
  constexpr int threads = 256;
  int blocks = (size + threads - 1) / threads;

  for(int i = 0; i < n_kernel; i++){
    // Static kernels
    shortKernel<<<blocks, threads, 0, s>>>(out_d, in_d, size, 1.004f);
    shortKernel<<<blocks, threads, 0, s>>>(in_d, out_d, size, 1.004f);

    // kernels with dynamic parameter `f`
    shortKernel<<<blocks, threads, 0, s>>>(out_d, in_d, size, f);
    shortKernel<<<blocks, threads, 0, s>>>(in_d, out_d, size, f);
  } 
}

void run_init(float *ptr, int size, float f, hipStream_t s) {
  constexpr int threads = 256;
  int blocks = (size + threads - 1) / threads;
  initKernel<<<blocks, threads, 0, s>>>(ptr, size, f);
}

int main() 
{
  gpu_graph_t _graph;
  gpu_graph_t _graph_always_recapture;

  _graph_always_recapture._always_recapture = true;

  // Set up memory, stream, events
  float *out_d = nullptr;
  float *in_d = nullptr;

  int size = 32;
  size_t bytes = size * sizeof(float);

  cudaErrCheck(hipMalloc(&out_d, bytes));
  cudaErrCheck(hipMalloc(&in_d, bytes));

  hipStream_t stream;
  hipEvent_t start, stop;
  cudaErrCheck(hipStreamCreate(&stream));
  cudaErrCheck(hipEventCreate(&start));
  cudaErrCheck(hipEventCreate(&stop));

  float scale = 1.0f;

  auto wrap_obj_graph = [&](gpu_graph_t &g, hipStream_t s) {
    run_kernels_graph(out_d, in_d, size, scale, g, s);
  };

  auto wrap_obj_no_graph = [&](gpu_graph_t &g, hipStream_t s) {
    run_kernels_no_graph(out_d, in_d, size, scale, stream);
  };

  run_init(out_d, size, 1.0f, stream);
  run_init(in_d, size, 1.0f, stream);

  printf("Running with    CUDA graph ('Recapture-then-update') ...\n");

  // Running the test with graph
  cudaErrCheck(hipEventRecord(start, stream));

  for(int i = 0; i < n_iteration; i++){
    scale = 1.0f + i * 0.001f;
    _graph_always_recapture.wrap(wrap_obj_no_graph, stream);
  }

  cudaErrCheck(hipEventRecord(stop, stream));
  cudaErrCheck(hipEventSynchronize(stop));

  float milliseconds;
  cudaErrCheck(hipEventElapsedTime(&milliseconds, start, stop));

  printf("Running with    CUDA graph ('Recapture-then-update') took %6.2f ms\n", milliseconds);

  run_init(out_d, size, 1.0f, stream);
  run_init(in_d, size, 1.0f, stream);

  printf("Running with    CUDA graph ('Combined Approach')     ...\n");

  // Running the test with graph
  cudaErrCheck(hipEventRecord(start, stream));

  for(int i = 0; i < n_iteration; i++){
    scale = 1.0f + i * 0.001f;
    _graph.wrap(wrap_obj_graph, stream);
  }

  cudaErrCheck(hipEventRecord(stop, stream));
  cudaErrCheck(hipEventSynchronize(stop));

  cudaErrCheck(hipEventElapsedTime(&milliseconds, start, stop));

  printf("Running with    CUDA graph ('Combined Approach')     took %6.2f ms\n", milliseconds);

  run_init(out_d, size, 1.0f, stream);
  run_init(in_d, size, 1.0f, stream);

  printf("Running without CUDA graph                           ...\n");

  cudaErrCheck(hipEventRecord(start, stream));

  for(int i = 0; i < n_iteration; i++){
    scale = 1.0f + i * 0.001f;
    run_kernels_no_graph(out_d, in_d, size, scale, stream);
  }

  cudaErrCheck(hipEventRecord(stop, stream));
  cudaErrCheck(hipEventSynchronize(stop));

  cudaErrCheck(hipEventElapsedTime(&milliseconds, start, stop));

  printf("Running without CUDA graph                           took %6.2f ms\n", milliseconds);

  // Finalize memory, stream, events
  cudaErrCheck(hipStreamDestroy(stream));
  cudaErrCheck(hipEventDestroy(start));
  cudaErrCheck(hipEventDestroy(stop));

  cudaErrCheck(hipFree(out_d));
  cudaErrCheck(hipFree(in_d));
}
